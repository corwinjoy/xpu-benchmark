#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample evaluates fair call and put prices for a
 * given set of European options by Black-Scholes formula.
 * See supplied whitepaper for more explanations.
 */

#include <chrono>

#include <helper_functions.h>  // helper functions for string parsing
#include <hip/hip_runtime_api.h>  // helper functions CUDA error checking and initialization

#include "number_with_commas.h"

////////////////////////////////////////////////////////////////////////////////
// Process an array of OptN options on GPU
////////////////////////////////////////////////////////////////////////////////
#include "American_kernel.cuh"

////////////////////////////////////////////////////////////////////////////////
// Process an array of optN options on CPU
////////////////////////////////////////////////////////////////////////////////
#include "BlackScholes_cpu.cpp"

////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high) {
    float t = (float) rand() / (float) RAND_MAX;
    return (1.0f - t) * low + t * high;
}

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
//const int OPT_N = 4000000;
//const int OPT_N = 6144*32;
const int OPT_N = 200;
const int NUM_ITERATIONS = 512;

const int OPT_SZ = OPT_N * sizeof(float);
const float RISKFREE = 0.02f;
const float VOLATILITY = 0.30f;

#define DIV_UP(a, b) (((a) + (b)-1) / (b))

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    // Start logs
    printf("[%s] - Starting...\n", argv[0]);

    //'h_' prefix - CPU (host) memory space
    float
    // Results calculated by CPU for reference
    *h_CallResultCPU,
    // CPU copy of GPU results
    *h_CallResultGPU,
    // CPU instance of input data
    *h_StockPrice, *h_OptionStrike, *h_OptionYears;

    //'d_' prefix - GPU (device) memory space
    float
    // Results calculated by GPU
    *d_CallResult,
    // GPU instance of input data
    *d_StockPrice, *d_OptionStrike, *d_OptionYears;

    double abs_pct_err, cpu, gpu, sum_pct_err, sum_cpu, max_pct_err, L1norm;

    StopWatchInterface *hTimer = NULL;
    int i;

    findCudaDevice(argc, (const char **) argv);

    sdkCreateTimer(&hTimer);

    printf("Initializing data...\n");
    printf("...allocating CPU memory for options.\n");
    h_CallResultCPU = (float *) malloc(OPT_SZ);
    h_CallResultGPU = (float *) malloc(OPT_SZ);
    h_StockPrice = (float *) malloc(OPT_SZ);
    h_OptionStrike = (float *) malloc(OPT_SZ);
    h_OptionYears = (float *) malloc(OPT_SZ);

    printf("...allocating GPU memory for options.\n");
    checkCudaErrors(hipMalloc((void **) &d_CallResult, OPT_SZ));
    checkCudaErrors(hipMalloc((void **) &d_StockPrice, OPT_SZ));
    checkCudaErrors(hipMalloc((void **) &d_OptionStrike, OPT_SZ));
    checkCudaErrors(hipMalloc((void **) &d_OptionYears, OPT_SZ));

    printf("...generating input data in CPU mem.\n");
    srand(5347);

    // Generate options set
    for (i = 0; i < OPT_N; i++) {
        h_CallResultCPU[i] = 0.0f;
        h_StockPrice[i] = RandFloat(5.0f, 30.0f);
        h_OptionStrike[i] = RandFloat(1.0f, 100.0f);
        h_OptionYears[i] = RandFloat(0.25f, 10.0f);
    }


    auto start = std::chrono::steady_clock::now();
    for (i = 0; i < NUM_ITERATIONS; i++) {
        // printf("...copying input data to GPU mem.\n");

        // Copy options data to GPU memory for further processing
        checkCudaErrors(
                hipMemcpy(d_StockPrice, h_StockPrice, OPT_SZ, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_OptionStrike, h_OptionStrike, OPT_SZ,
                                   hipMemcpyHostToDevice));
        checkCudaErrors(
                hipMemcpy(d_OptionYears, h_OptionYears, OPT_SZ, hipMemcpyHostToDevice));
        // printf("Data init done.\n\n");

        // printf("Executing American Option GPU kernel (%i iterations)...\n", NUM_ITERATIONS);
        checkCudaErrors(hipDeviceSynchronize());


        AmerGPU<<<DIV_UP((OPT_N / 2), 128), 16>>>(
                (float2 *) d_CallResult, (float2 *) d_StockPrice,
                (float2 *) d_OptionStrike, (float2 *) d_OptionYears, RISKFREE, VOLATILITY,
                OPT_N);
        getLastCudaError("AmerGPU() execution failed\n");


        // printf("\nReading back GPU results...\n");
        // Read back GPU results to compare them to CPU results
        checkCudaErrors(hipMemcpy(h_CallResultGPU, d_CallResult, OPT_SZ,
                                   hipMemcpyDeviceToHost));

        checkCudaErrors(hipDeviceSynchronize());
    }

    auto stop = std::chrono::steady_clock::now();
    auto duration_ms = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();
    duration_ms /= NUM_ITERATIONS;
    double duration_s = duration_ms / pow(10, 6);


    // Both call and put is calculated
    std::cout << "Options Count = " << numberFormatWithCommas(OPT_N) << " took " << numberFormatWithCommas(duration_ms)
              << " microseconds";
    std::cout << ", or " << duration_s << " seconds" << std::endl;

    printf("Checking the results...\n");
    printf("...running CPU calculations.\n\n");
    // Calculate options values on CPU
    BlackScholesCPU(h_CallResultCPU, h_StockPrice, h_OptionStrike,
                    h_OptionYears, RISKFREE, VOLATILITY, OPT_N);

    printf("Comparing the results...\n");
    // Calculate max absolute difference and L1 distance
    // between CPU and GPU results
    sum_pct_err = 0;
    sum_cpu = 0;
    max_pct_err = 0;

    for (i = 0; i < OPT_N; i++) {
        cpu = h_CallResultCPU[i];
        gpu = h_CallResultGPU[i];
        abs_pct_err = fabs((cpu - gpu)/(cpu+0.0001));

        if (abs_pct_err > max_pct_err) {
            max_pct_err = abs_pct_err;
        }

        sum_pct_err += abs_pct_err;
        sum_cpu += fabs(cpu);
    }

    L1norm = sum_pct_err / (double)OPT_N;
    printf("L1 norm: %E\n", L1norm);
    printf("Max absolute pct error: %E\n\n", max_pct_err);

    printf("Shutting down...\n");
    printf("...releasing GPU memory.\n");
    checkCudaErrors(hipFree(d_OptionYears));
    checkCudaErrors(hipFree(d_OptionStrike));
    checkCudaErrors(hipFree(d_StockPrice));
    checkCudaErrors(hipFree(d_CallResult));

    printf("...releasing CPU memory.\n");
    free(h_OptionYears);
    free(h_OptionStrike);
    free(h_StockPrice);
    free(h_CallResultGPU);
    free(h_CallResultCPU);
    sdkDeleteTimer(&hTimer);
    printf("Shutdown done.\n");

    printf("\n[BlackScholes] - Test Summary\n");

    if (L1norm > 1e-6) {
        printf("Test failed!\n");
        exit(EXIT_FAILURE);
    }

    printf("Test passed\n");
    exit(EXIT_SUCCESS);
}
